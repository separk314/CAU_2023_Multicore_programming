#include "hip/hip_runtime.h"

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>

#define NUM_STEPS 200000
#define STEP 1.0/NUM_STEPS

struct calculation {
  __host__ __device__
  double operator()(double i) {
    double x = (i+0.5)*STEP;
    return 4.0/(1.0+x*x);
  }
};

int main ()
{ 
	clock_t start_time = clock();

  // value "i" initialization
  thrust::device_vector<double> i(NUM_STEPS); // same with "i" in the original code
  thrust::sequence(i.begin(), i.end());    // 0 to NUM_STEPS in "i" vector

  thrust::device_vector<double> sum(NUM_STEPS); // vector to store the "sum"
  // same with the "for" statement in the original code
  thrust::transform(i.begin(), i.end(), sum.begin(), calculation());    

  double result = thrust::reduce(sum.begin(), sum.end());    // summation
  double pi = STEP * result;

  // Calulate the total execution time
	clock_t end_time = clock();
  clock_t exe_time = end_time - start_time;
  double exe_time_sec = (double)(exe_time) / CLOCKS_PER_SEC;
  printf("Execution Time : %.10lf sec \n", exe_time_sec);

	printf("pi = %.10lf \n",pi);
}
