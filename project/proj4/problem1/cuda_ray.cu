

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048


// Indicates 3D sphere shape: sphere location, radius, and color information
struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
};

__global__ void CUDA_kernel(Sphere* s, unsigned char* ptr) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y*DIM;
  float ox = (x - DIM/2);
	float oy = (y - DIM/2);

  float r=0, g=0, b=0;
	float   maxz = -INF;

	for(int i=0; i<SPHERES; i++) {
		float dx = ox - s[i].x;
    float dy = oy - s[i].y;
		float t, n;

		// hit() function
    if (dx * dx + dy * dy < s[i].radius * s[i].radius) {
    	float dz = sqrtf(s[i].radius * s[i].radius - dx * dx - dy * dy);
    	n = dz / sqrtf(s[i].radius * s[i].radius);
    	t = dz + s[i].z;
    } else {
			t = -INF;
		}

		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}

	ptr[offset*4 + 0] = (int)(r * 255);
	ptr[offset*4 + 1] = (int)(g * 255);
	ptr[offset*4 + 2] = (int)(b * 255);
	ptr[offset*4 + 3] = 255;
}

// Function to store images in PPM file format
void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp)
{
	int i,x,y;
	fprintf(fp,"P3\n");
	fprintf(fp,"%d %d\n",xdim, ydim);
	fprintf(fp,"255\n");
	for (y=0;y<ydim;y++) {
		for (x=0;x<xdim;x++) {
			i=x+y*xdim;
			fprintf(fp,"%d %d %d ",bitmap[4*i],bitmap[4*i+1],bitmap[4*i+2]);
		}
		fprintf(fp,"\n");
	}
	printf("[result.ppm] was generated. \n");
}

int main(int argc, char* argv[])
{
	srand(time(NULL));
  FILE* fp = fopen("result.ppm", "w");

	// temp_s: Sphere used by the CPU
	Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
	for (int i=0; i<SPHERES; i++) {
		temp_s[i].r = rnd( 1.0f );
		temp_s[i].g = rnd( 1.0f );
		temp_s[i].b = rnd( 1.0f );
		temp_s[i].x = rnd( 2000.0f ) - 1000;
		temp_s[i].y = rnd( 2000.0f ) - 1000;
		temp_s[i].z = rnd( 2000.0f ) - 1000;
		temp_s[i].radius = rnd( 200.0f ) + 40;
	}

	// cuda_s: Sphere used by the GPU
	Sphere *cuda_s;
	hipMalloc((void**)&cuda_s, sizeof(Sphere) * SPHERES);
	hipMemcpy(cuda_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

	// bitmap: Bitmap used by CPU
	unsigned char* bitmap;
	bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM*DIM*4);

	// cuda_bitmap: Bitmap used by GPU
	unsigned char* cuda_bitmap;
	hipMalloc((void**)&cuda_bitmap, sizeof(unsigned char) *DIM*DIM*4);
	hipMemcpy(cuda_bitmap, bitmap, sizeof(unsigned char)*DIM*DIM*4, hipMemcpyHostToDevice);

  dim3 gridDims(DIM / 16, DIM / 16);
  dim3 blockDims(16, 16);

	clock_t start = clock();
  CUDA_kernel<<<gridDims, blockDims>>>(cuda_s, cuda_bitmap);
	clock_t end = clock();

  hipDeviceSynchronize();
	hipMemcpy(bitmap, cuda_bitmap, sizeof(unsigned char)*DIM*DIM*4, hipMemcpyDeviceToHost);
	
	clock_t exe_time = end - start;
	double exe_time_ms = ((double)exe_time / CLOCKS_PER_SEC) * 1000.0;
	printf("CUDA ray tracing: %f ms \n", exe_time_ms);

	ppm_write(bitmap,DIM,DIM,fp);

	fclose(fp);
	free(bitmap);
	free(temp_s);
	free(cuda_bitmap);
	free(cuda_s);

	return 0;
}
